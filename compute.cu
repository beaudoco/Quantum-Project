#include "hip/hip_runtime.h"
// compute.cu
//
// driver and kernel call

#include <stdio.h>   // for printf
#include <stdlib.h>  // for malloc
#include <complex.h> // for double complex
#include <string.h>  // for memcpy()
#include <unistd.h>  // for sleep()
#include <math.h>    // for pow()
#include <stdbool.h> // for bool

#define THREADS_PER_BLOCK 512

typedef struct Qureg
{

    int rank;
    int numRanks;

    int numQubits;
    long long int numAmpsTotal;
    long long int numAmpsPerRank;

    double complex *stateVector;
    double complex *bufferVector;
} Qureg;
 
// __global__ void compute_d (double complex *a_d, double complex *b_d, int n)
__global__ void compute_d (double complex a_d, int n)

{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    if (x < n) {
        a_d[x] = a_d[x] * 2;
	}
		
}

extern "C" void matrixMultiplication(Qureg qureg, Qureg *qureg2, int arraySize)
{
    double complex a_d;
    // Qureg *b_d;    

	// int *a_d, *b_d, *c_d;

	hipMalloc ((void**) a_d, sizeof(double complex) * arraySize);
	// hipMalloc ((void**) &b_d, sizeof(Qureg) * arraySize);
	// hipMalloc ((void**) &c_d, sizeof(int) * arraySize);

    // compute_d <<< ceil((float) arraySize/THREADS_PER_BLOCK), THREADS_PER_BLOCK >>> (a_d, b_d, arraySize);
    compute_d <<< ceil((float) arraySize/THREADS_PER_BLOCK), THREADS_PER_BLOCK >>> (a_d, arraySize);
	
	hipError_t err = hipGetLastError();
	if (err != hipSuccess)
		printf ("CUDA error: %s\n", hipGetErrorString(err));
		
	hipMemcpy (qureg.stateVector, a_d, sizeof(Qureg) * arraySize, hipMemcpyDeviceToHost);
	hipFree (a_d);
	// hipFree (b_d);
	// hipFree (c_d);
}
